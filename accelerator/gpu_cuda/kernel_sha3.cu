#include "hip/hip_runtime.h"
// sha3.c
// 19-Nov-11  Markku-Juhani O. Saarinen <mjos@iki.fi>

// Revised 07-Aug-15 to match with official release of FIPS PUB 202 "SHA3"
// Revised 03-Sep-15 for portability + OpenSSL - style API

#include <stdio.h>
#include <stdint.h>
#include "header_gpu.h"
#include "cuda_util.h"

#define TPB 512

#ifndef KECCAKF_ROUNDS
#define KECCAKF_ROUNDS 24
#endif

#ifndef ROTL64
#define ROTL64(x, y) (((x) << (y)) | ((x) >> (64 - (y))))
#endif

// state context
typedef struct
{
    union
    {                   // state:
        uint8_t b[200]; // 8-bit bytes
        uint64_t q[25]; // 64-bit words
    } st;
    int pt, rsiz, mdlen; // these don't overflow
} sha3_ctx_t;

// constants
const uint64_t host_keccakf_rndc[24] = {
    0x0000000000000001, 0x0000000000008082, 0x800000000000808a,
    0x8000000080008000, 0x000000000000808b, 0x0000000080000001,
    0x8000000080008081, 0x8000000000008009, 0x000000000000008a,
    0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
    0x000000008000808b, 0x800000000000008b, 0x8000000000008089,
    0x8000000000008003, 0x8000000000008002, 0x8000000000000080,
    0x000000000000800a, 0x800000008000000a, 0x8000000080008081,
    0x8000000000008080, 0x0000000080000001, 0x8000000080008008};
const int host_keccakf_rotc[24] = {
    1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 2, 14,
    27, 41, 56, 8, 25, 43, 62, 18, 39, 61, 20, 44};
const int host_keccakf_piln[24] = {
    10, 7, 11, 17, 18, 3, 5, 16, 8, 21, 24, 4,
    15, 23, 19, 13, 12, 2, 20, 14, 22, 9, 6, 1};

__device__ __constant__ uint64_t gpu_keccakf_rndc[24];
__device__ __constant__ int gpu_keccakf_rotc[24];
__device__ __constant__ int gpu_keccakf_piln[24];

sha3_ctx_t *gpu_contexts;
char *gpu_in;
int *gpu_inlen;
int *gpu_offset;
char *gpu_md;

int *offsets;
char *out;

// update the state with given number of rounds

__device__ void sha3_keccakf(uint64_t st[25])
{
    // variables
    int i, j, r;
    uint64_t t, bc[5];

#if __BYTE_ORDER__ != __ORDER_LITTLE_ENDIAN__
    uint8_t *v;

    // endianess conversion. this is redundant on little-endian targets
    for (i = 0; i < 25; i++)
    {
        v = (uint8_t *)&st[i];
        st[i] = ((uint64_t)v[0]) | (((uint64_t)v[1]) << 8) |
                (((uint64_t)v[2]) << 16) | (((uint64_t)v[3]) << 24) |
                (((uint64_t)v[4]) << 32) | (((uint64_t)v[5]) << 40) |
                (((uint64_t)v[6]) << 48) | (((uint64_t)v[7]) << 56);
    }
#endif

    // actual iteration
    for (r = 0; r < KECCAKF_ROUNDS; r++)
    {

        // Theta
        for (i = 0; i < 5; i++)
            bc[i] = st[i] ^ st[i + 5] ^ st[i + 10] ^ st[i + 15] ^ st[i + 20];

        for (i = 0; i < 5; i++)
        {
            t = bc[(i + 4) % 5] ^ ROTL64(bc[(i + 1) % 5], 1);
            for (j = 0; j < 25; j += 5)
                st[j + i] ^= t;
        }

        // Rho Pi
        t = st[1];
        for (i = 0; i < 24; i++)
        {
            j = gpu_keccakf_piln[i];
            bc[0] = st[j];
            st[j] = ROTL64(t, gpu_keccakf_rotc[i]);
            t = bc[0];
        }

        //  Chi
        for (j = 0; j < 25; j += 5)
        {
            for (i = 0; i < 5; i++)
                bc[i] = st[j + i];
            for (i = 0; i < 5; i++)
                st[j + i] ^= (~bc[(i + 1) % 5]) & bc[(i + 2) % 5];
        }

        //  Iota
        st[0] ^= gpu_keccakf_rndc[r];
    }

#if __BYTE_ORDER__ != __ORDER_LITTLE_ENDIAN__
    // endianess conversion. this is redundant on little-endian targets
    for (i = 0; i < 25; i++)
    {
        v = (uint8_t *)&st[i];
        t = st[i];
        v[0] = t & 0xFF;
        v[1] = (t >> 8) & 0xFF;
        v[2] = (t >> 16) & 0xFF;
        v[3] = (t >> 24) & 0xFF;
        v[4] = (t >> 32) & 0xFF;
        v[5] = (t >> 40) & 0xFF;
        v[6] = (t >> 48) & 0xFF;
        v[7] = (t >> 56) & 0xFF;
    }
#endif
}

// Initialize the context for SHA3

__device__ void sha3_init(sha3_ctx_t *c, int mdlen)
{
    int i;

    for (i = 0; i < 25; i++)
        c->st.q[i] = 0;
    c->mdlen = mdlen;
    c->rsiz = 200 - 2 * mdlen;
    c->pt = 0;
}

// update state with more data

__device__ void sha3_update(sha3_ctx_t *c, const void *data, size_t len)
{
    size_t i;
    int j;

    j = c->pt;
    for (i = 0; i < len; i++)
    {
        c->st.b[j++] ^= ((const uint8_t *)data)[i];
        if (j >= c->rsiz)
        {
            sha3_keccakf(c->st.q);
            j = 0;
        }
    }
    c->pt = j;
}

// finalize and output a hash
__device__ void sha3_final(void *md, sha3_ctx_t *c)
{
    int i;

    c->st.b[c->pt] ^= 0x06;
    c->st.b[c->rsiz - 1] ^= 0x80;
    sha3_keccakf(c->st.q);

    for (i = 0; i < c->mdlen; i++)
    {
        ((uint8_t *)md)[i] = c->st.b[i];
    }
}

// compute a SHA-3 hash (md) of given byte length from "in"

__global__ void sha3(int num, sha3_ctx_t *contexts, const void *in, int *inlen, int *offset, void *md, int mdlen)
{
    // int const tid = threadIdx.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    sha3_init(contexts + tid, mdlen);
    sha3_update(contexts + tid, in + offset[tid], inlen[tid]);
    sha3_final(md + (tid * mdlen), contexts + tid);
}

__host__ void init_gpu_keccak(int num)
{
    CHECKCUDAERR(hipMemcpyToSymbol(HIP_SYMBOL(gpu_keccakf_rndc), host_keccakf_rndc, 24 * sizeof(uint64_t)));
    CHECKCUDAERR(hipMemcpyToSymbol(HIP_SYMBOL(gpu_keccakf_rotc), host_keccakf_rotc, 24 * sizeof(int)));
    CHECKCUDAERR(hipMemcpyToSymbol(HIP_SYMBOL(gpu_keccakf_piln), host_keccakf_piln, 24 * sizeof(int)));

    CHECKCUDAERR(hipMalloc(&gpu_contexts, num * sizeof(sha3_ctx_t)));
    CHECKCUDAERR(hipMalloc(&gpu_in, num * 1024));
    CHECKCUDAERR(hipMalloc(&gpu_inlen, num * sizeof(int)));
    CHECKCUDAERR(hipMalloc(&gpu_offset, num * sizeof(int)));
    CHECKCUDAERR(hipMalloc(&gpu_md, num * 32));
    offsets = (int *)malloc(num * sizeof(int));
    if (!offsets)
    {
        printf("Error in allocating CPU offsets!\n");
    }

    out = (char *)malloc(num * 32);
    if (!offsets)
    {
        printf("Error in allocating CPU output!\n");
    }
}

unsigned char *run_keccak(int batch_size, unsigned char *messages, int *message_lengths)
{
    // v1: variable offsets
    // int sum = message_lengths[0];
    // offsets[0] = 0;
    // for (int i = 1; i < batch_size; i++) {
    // 	offsets[i] = sum;
    // 	sum += message_lengths[i];
    // }

    // v2: alligned at 1024 bytes
    for (int i = 0; i < batch_size; i++)
    {
        offsets[i] = i * 1024;
    }
    int sum = 1024 * batch_size;

    CHECKCUDAERR(hipMemcpy(gpu_in, messages, sum, hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(gpu_inlen, message_lengths, batch_size * sizeof(int), hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(gpu_offset, offsets, batch_size * sizeof(int), hipMemcpyHostToDevice));

    // sha3<<<1, batch_size>>>(batch_size, gpu_contexts, gpu_in, gpu_inlen, gpu_offset, gpu_md, 32);
    sha3<<<batch_size / TPB, TPB>>>(batch_size, gpu_contexts, gpu_in, gpu_inlen, gpu_offset, gpu_md, 32);

    CHECKCUDAERR(hipMemcpy(out, gpu_md, batch_size * 32, hipMemcpyDeviceToHost));

    return (unsigned char *)out;
}

void free_gpu_keccak()
{
    hipFree(gpu_contexts);
    hipFree(gpu_in);
    hipFree(gpu_inlen);
    hipFree(gpu_offset);
    hipFree(gpu_md);
    free(offsets);
    free(out);
}
